#include "hip/hip_runtime.h"
//
// Created by Hervé Paulino on 27/09/2020.
//

#include <gtest/gtest.h>
#include <fstream>
#include <algorithm>

#include <nbody/cuda_nbody_all_pairs.h>
// #include <nbody/cuda_nbody_gmem_no_cycles.h>
// #include <nbody/cuda_nbody_smem_no_cycles.h>
// #include <nbody/cuda_nbody_first.h>

/**
 * Compares the result produced and stored in stream result_stream with the log in file logfilename
 *
 * @param result_stream
 * @param filename
 */
void compare_results(std::stringstream& result_stream, std::string& logfilename) {
    std::string bufferResult;
    std::string bufferExpected;
    std::ifstream ins(logfilename);
    ASSERT_TRUE(ins.is_open());

    while (getline (result_stream, bufferResult)) {
        getline (ins, bufferExpected);
        std::string expected;
        expected.reserve(bufferExpected.size());
        std::copy_if(bufferExpected.begin(), bufferExpected.end(), std::back_inserter(expected),
                     [] (char c) { return c != '\r'; });

        EXPECT_STREQ(bufferResult.c_str(), expected.c_str());
    }

    ins.close();
}


TEST(NBody, CUDA_All_Pairs_P1000_T10_U0_T4) {

    auto nparticles = 1000;
    auto T_FINAL = 10.0;
    auto universe = cadlabs::universe_t::ORIGINAL;
    auto number_of_threads = 2;
    auto number_of_streams = 10;
    std::string original_result_log = "p1000_t10_u0.log";

    cadlabs::cuda_nbody_all_pairs nbody(nparticles, T_FINAL, number_of_threads, universe, 0, "", 256, number_of_streams);
    nbody.run_simulation();

    std::stringstream ss;
    nbody.print_all_particles(ss);
    compare_results(ss, original_result_log);
}
