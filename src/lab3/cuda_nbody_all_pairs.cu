#include "hip/hip_runtime.h"
/**
 * Hervé Paulino
 */

#include <nbody/cuda_nbody_all_pairs.h>
#include <omp.h>
#include "stdio.h"

static constexpr int thread_block_size = 32;

namespace cadlabs {

cuda_nbody_all_pairs::cuda_nbody_all_pairs(
        const int number_particles,
        const float t_final,
        const unsigned number_of_threads,
        const universe_t universe,
        const unsigned universe_seed,
        const string file_name) :
        nbody(number_particles, t_final, universe, universe_seed, file_name),
        number_blocks ((number_particles + thread_block_size - 1)/thread_block_size)  {

    // hipMalloc((void **)&gpu_particles, number_particles*sizeof(particle_t));
}

cuda_nbody_all_pairs::~cuda_nbody_all_pairs() {
    // hipFree(gpu_particles);
}

__global__ void two_cycles_parallel(particle_t* particles, const unsigned number_particles) {
    int targetParticle = blockIdx.x * blockDim.x + threadIdx.x;
    int forceEffectParticle = blockIdx.y * blockDim.y + threadIdx.y;
    if (targetParticle < number_particles && forceEffectParticle < number_particles) {
        particle_t *tp = &particles[targetParticle];
        particle_t *fp = &particles[forceEffectParticle];
        double x_sep = fp->x_pos - tp->x_pos;
        double y_sep = fp->y_pos - tp->y_pos;
        double dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);
        double grav_base = GRAV_CONSTANT * (fp->mass) * (tp->mass) / dist_sq;
        atomicAdd(&(tp->x_force), grav_base * x_sep);
        atomicAdd(&(tp->y_force), grav_base * y_sep);
    }
}

/**
 * TODO: A CUDA implementation
 */
void cuda_nbody_all_pairs::calculate_forces() {
    hipMalloc((void **)&gpu_particles, number_particles*sizeof(particle_t));
    uint count = number_particles * sizeof(particle_t);

    /*
     * Setting the forces to 0 within a kernel would require the synchronization of all blocks
     * An alternative solution to using the host would be to launch a kernel specifically to
     *  set the forces to 0. However, the number of particles will either not be high enough to warrant
     *  launching a kernel, or will be so high that the time to compute the forces between the
     *  particles completely eclipses the time required to set the forces to 0.
     */
    #pragma omp parallel for num_threads(number_of_threads)
    for(int i = 0; i < number_particles; i++) {
        particle_t* p = &particles[i];
        p->x_force = 0;
        p->y_force = 0;
    }

    hipMemcpy(gpu_particles, particles, count, hipMemcpyHostToDevice);
    dim3 grid(number_blocks, number_blocks);
    dim3 block(thread_block_size, thread_block_size);
    two_cycles_parallel<<<grid, block>>>(gpu_particles, number_particles);
    hipMemcpy(particles, gpu_particles, count, hipMemcpyDeviceToHost);
    hipFree(gpu_particles);
}


void cuda_nbody_all_pairs::move_all_particles(double step) {
    nbody::move_all_particles(step);
}

void cuda_nbody_all_pairs::print_all_particles(std::ostream &out) {
    nbody::print_all_particles(out);
}


} // namespace

