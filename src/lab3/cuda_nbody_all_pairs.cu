#include "hip/hip_runtime.h"
#include <nbody/cuda_nbody_all_pairs.h>
#include <omp.h>
#include <stdio.h>


// static constexpr int BLOCK_HEIGHT = 2;
//constexpr uint numStreams = 1;

namespace cadlabs {

    cuda_nbody_all_pairs::cuda_nbody_all_pairs(
            const int number_particles,
            const float t_final,
            const unsigned n,
            const universe_t universe,
            const unsigned universe_seed,
            const string file_name,
            const int blockWidth,
            const int blockHeight,
            const int n_streams) :
            nbody(number_particles, t_final, universe, universe_seed, file_name),
            blockWidth(blockWidth), n(n), numStreams(n_streams), blockHeight(blockHeight) {


#ifdef SOA
        hipMalloc((void **)&gpu_particles_soa.x_pos, number_particles*sizeof(double));
        hipMalloc((void **)&gpu_particles_soa.y_pos, number_particles*sizeof(double));

        hipMalloc((void **)&gpu_particles_soa.x_vel, number_particles*sizeof(double));
        hipMalloc((void **)&gpu_particles_soa.y_vel, number_particles*sizeof(double));

        hipMalloc((void **)&gpu_particles_soa.x_force, number_particles*sizeof(double));
        hipMalloc((void **)&gpu_particles_soa.y_force, number_particles*sizeof(double));

        hipMalloc((void **)&gpu_particles_soa.mass, number_particles*sizeof(double));

        // We can do this because
        // the mass of the particles
        // stays constant across the
        // whole program
        hipMemcpy(gpu_particles_soa.mass, particles_soa.mass,
                   number_particles * sizeof(double), hipMemcpyHostToDevice);
#else
        hipMalloc((void **)&gpu_particles, number_particles*sizeof(particle_t));
#endif
        hipMalloc(&gpu_particles, number_particles*sizeof(particle_t));
        gridWidth  = number_particles / (blockWidth * 2 * n) + (number_particles % (blockWidth * 2 * n) != 0);
        gridHeight = number_particles / (blockHeight) + (number_particles % (blockHeight) != 0);

        hipHostMalloc(&hForcesX, number_particles * gridWidth * sizeof(double));
        hipHostMalloc(&hForcesY, number_particles * gridWidth * sizeof(double));

        hipMalloc(&dForcesX, number_particles * gridWidth * sizeof(double));
        hipMalloc(&dForcesY, number_particles * gridWidth * sizeof(double));
    }

    cuda_nbody_all_pairs::~cuda_nbody_all_pairs() {
#ifdef SOA
        hipFree(gpu_particles_soa.x_pos);
        hipFree(gpu_particles_soa.y_pos);

        hipFree(gpu_particles_soa.x_vel);
        hipFree(gpu_particles_soa.y_vel);

        hipFree(gpu_particles_soa.x_force);
        hipFree(gpu_particles_soa.y_force);

        hipFree(gpu_particles_soa.mass);
#else
        hipFree(gpu_particles);
#endif
        hipHostFree(hForcesX);
        hipHostFree(hForcesY);
        hipFree(dForcesX);
        hipFree(dForcesY);
    }


    //STREAM IMPLEMENTATION WITH ARRAYS OF STRUCTURES
    template<unsigned int blockSize, unsigned int blockHeight>
    __global__ void calculate_forces_two_cycles_parallel(particle_t * __restrict__ particles, const unsigned int targetOffset,
                                     double * __restrict__ gForcesX, double * __restrict__ gForcesY,
                                     const unsigned int number_particles,
                                     const unsigned int gridWidth, const unsigned int n) {

        __shared__ double sForcesX[blockHeight * blockSize];
        __shared__ double sForcesY[blockHeight * blockSize];

        unsigned int forceParticle  = blockIdx.x * 2 * blockDim.x + threadIdx.x;
        unsigned int targetParticle = blockIdx.y * blockDim.y + threadIdx.y + targetOffset;
        unsigned int gridSize = blockDim.x * 2 * gridDim.x, i = 0;

        sForcesX[threadIdx.y * blockDim.x + threadIdx.x] = .0;
        sForcesY[threadIdx.y * blockDim.x + threadIdx.x] = .0;

        if (forceParticle < number_particles
            && targetParticle < number_particles) {
            /*
             * Mapping section
             */

            while (i < n) {
                int a = (forceParticle < number_particles);
                int b = ((forceParticle + blockDim.x) < number_particles);

                particle_t *fp_1 = &particles[forceParticle], *fp_2 = &particles[forceParticle + blockDim.x];
                particle_t *tp = &particles[targetParticle];

                double x_sep_1 = fp_1->x_pos - tp->x_pos, x_sep_2 = fp_2->x_pos - tp->x_pos;
                double y_sep_1 = fp_1->y_pos - tp->y_pos, y_sep_2 = fp_2->y_pos - tp->y_pos;

                double dist_sq_1 = MAX((x_sep_1 * x_sep_1) + (y_sep_1 * y_sep_1), 0.01);
                double dist_sq_2 = MAX((x_sep_2 * x_sep_2) + (y_sep_2 * y_sep_2), 0.01);

                double grav_base_1 = GRAV_CONSTANT * (fp_1->mass) * (tp->mass) / dist_sq_1;
                double grav_base_2 = GRAV_CONSTANT * (fp_2->mass) * (tp->mass) / dist_sq_2;

                sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                        a * (grav_base_1 * x_sep_1) + b * (grav_base_2 * x_sep_2);
                sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                        a * (grav_base_1 * y_sep_1) + b * (grav_base_2 * y_sep_2);

                forceParticle += gridSize;
                i++;
            }
            __syncthreads();

            /*
             * Reduce section
             */
            if (blockSize >= 512) {
                if (threadIdx.x < 256) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 256];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 256];
                }
                __syncthreads();
            }

            if (blockSize >= 256) {
                if (threadIdx.x < 128) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 128];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 128];
                }
                __syncthreads();
            }

            if (blockSize >= 128) {
                if (threadIdx.x < 64) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 64];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 64];
                }
                __syncthreads();
            }

            unsigned int s = blockDim.x / 2;

            if (blockSize >= 512) s >>= 3;
            else if (blockSize >= 256) s >>= 2;
            else if (blockSize >= 128) s >>= 1;

            if (threadIdx.x < s) {
                if (blockSize >= 64) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 32];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 32];
                    s >>= 1;
                }

                if (blockSize >= 32) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 16) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 8) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 4) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 2) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                }
            }

            if (!threadIdx.x) {
                gForcesX[targetParticle * gridWidth + blockIdx.x] = sForcesX[threadIdx.y * blockDim.x];
                gForcesY[targetParticle * gridWidth + blockIdx.x] = sForcesY[threadIdx.y * blockDim.x];
            }
        }
    }

    template<unsigned int blockSize>
    __global__ void calculate_forces_two_cycles_parallel_soa(
            const double * __restrict__ x_pos, const double * __restrict__ y_pos,
            const double * __restrict__ mass, const unsigned int target_offset,
            double * __restrict__ gForcesX, double * __restrict__ gForcesY,
            const unsigned int number_particles,
            const unsigned int gridWidth, const unsigned int n) {

        __shared__ double sForcesX[1 * blockSize];
        __shared__ double sForcesY[1 * blockSize];

        unsigned int forceParticle  = blockIdx.x * 2 * blockDim.x + threadIdx.x;
        unsigned int targetParticle = blockIdx.y * blockDim.y + threadIdx.y + target_offset;
        unsigned int gridSize = blockDim.x * 2 * gridDim.x, i = 0;

        sForcesX[threadIdx.y * blockDim.x + threadIdx.x] = .0;
        sForcesY[threadIdx.y * blockDim.x + threadIdx.x] = .0;

        if (forceParticle < number_particles
            && targetParticle < number_particles) {

            /*
             * Mapping section
             */
            while (i < n) {
                int a = (forceParticle < number_particles);
                int b = ((forceParticle + blockDim.x) < number_particles);

                double x_sep_1 = x_pos[forceParticle] - x_pos[targetParticle],
                x_sep_2 = x_pos[forceParticle + blockDim.x] - x_pos[targetParticle];
                double y_sep_1 = y_pos[forceParticle] - y_pos[targetParticle],
                y_sep_2 = y_pos[forceParticle + blockDim.x] - y_pos[targetParticle];

                double dist_sq_1 = MAX((x_sep_1 * x_sep_1) + (y_sep_1 * y_sep_1), 0.01);
                double dist_sq_2 = MAX((x_sep_2 * x_sep_2) + (y_sep_2 * y_sep_2), 0.01);

                double grav_base_1 = GRAV_CONSTANT * (mass[forceParticle])
                        * (mass[targetParticle]) / dist_sq_1;
                double grav_base_2 = GRAV_CONSTANT * (mass[forceParticle + blockDim.x])
                        * (mass[targetParticle]) / dist_sq_2;

                sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                        a * (grav_base_1 * x_sep_1) + b * (grav_base_2 * x_sep_2);
                sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                        a * (grav_base_1 * y_sep_1) + b * (grav_base_2 * y_sep_2);

                forceParticle += gridSize;
                i++;
            }
            __syncthreads();

            /*
             * Reduce section
             */
            if (blockSize >= 512) {
                if (threadIdx.x < 256) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 256];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 256];
                }
                __syncthreads();
            }

            if (blockSize >= 256) {
                if (threadIdx.x < 128) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 128];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 128];
                }
                __syncthreads();
            }

            if (blockSize >= 128) {
                if (threadIdx.x < 64) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 64];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 64];
                }
                __syncthreads();
            }

            unsigned int s = blockDim.x / 2;

            if (blockSize >= 512) s >>= 3;
            else if (blockSize >= 256) s >>= 2;
            else if (blockSize >= 128) s >>= 1;

            if (threadIdx.x < s) {
                if (blockSize >= 64) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + 32];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + 32];
                    s >>= 1;
                }

                if (blockSize >= 32) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 16) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 8) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 4) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                    s >>= 1;
                }

                if (blockSize >= 2) {
                    sForcesX[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesX[threadIdx.y * blockDim.x + threadIdx.x + s];
                    sForcesY[threadIdx.y * blockDim.x + threadIdx.x] +=
                            sForcesY[threadIdx.y * blockDim.x + threadIdx.x + s];
                }
            }

            if (!threadIdx.x) {
                gForcesX[targetParticle * gridWidth + blockIdx.x] = sForcesX[threadIdx.y * blockDim.x];
                gForcesY[targetParticle * gridWidth + blockIdx.x] = sForcesY[threadIdx.y * blockDim.x];
            }
        }
    }

#ifdef SOA
    // Having this in a separate method for this
    // might lead to a small performance loss
    static void call_kernel_soa(
            int block_width,
            const double * x_pos, const double * y_pos,
            const double * mass, const int target_offset,
            double * gForcesX, double * gForcesY,
            const unsigned int number_particles,
            const unsigned int gridWidth, const unsigned int n, dim3 grid, dim3 block) {

        switch (block_width) {
            case 1024:
                calculate_forces_two_cycles_parallel_soa<1024><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                            number_particles, gridWidth, n);
                break;
            case 512:
                calculate_forces_two_cycles_parallel_soa<512><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                           number_particles, gridWidth, n);
                break;
            case 256:
                calculate_forces_two_cycles_parallel_soa<256><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                           number_particles, gridWidth, n);
                break;
            case 128:
                calculate_forces_two_cycles_parallel_soa<128><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                           number_particles, gridWidth, n);
                break;
            case 64:
                calculate_forces_two_cycles_parallel_soa<64><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                          number_particles, gridWidth, n);
                break;
            case 32:
                calculate_forces_two_cycles_parallel_soa<32><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                          number_particles, gridWidth, n);
                break;
            case 16:
                calculate_forces_two_cycles_parallel_soa<16><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                          number_particles, gridWidth, n);
                break;
            case 8:
                calculate_forces_two_cycles_parallel_soa<8><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                         number_particles, gridWidth, n);
                break;
            case 4:
                calculate_forces_two_cycles_parallel_soa<4><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                         number_particles, gridWidth, n);
                break;
            case 2:
                calculate_forces_two_cycles_parallel_soa<2><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                         number_particles, gridWidth, n);
                break;
            case 1:
                calculate_forces_two_cycles_parallel_soa<1><<<grid, block>>>(x_pos, y_pos, mass, target_offset, gForcesX, gForcesY,
                                                                         number_particles, gridWidth, n);
                break;
        }
    }

#else

    template <unsigned int block_width>
    static void call_kernel(
            int block_height, particle_t *particles, int targetOffset, double *gForcesX,
            double *gForcesY, const unsigned int number_particles,
            const unsigned int gridWidth, const unsigned int n, dim3 grid, dim3 block, hipStream_t stream) {

        if (block_width <= 1) {
            if (block_height == 1024) {
                calculate_forces_two_cycles_parallel<block_width, 1024><<<grid, block, 0, stream>>>(particles,
                                                                                                    targetOffset,
                                                                                                    gForcesX,
                                                                                                    gForcesY,
                                                                                                    number_particles,
                                                                                                    gridWidth, n);
                return;
            }
        }

        if (block_width <= 2) {
            if(block_height == 512) {
                calculate_forces_two_cycles_parallel<block_width, 512><<<grid, block, 0, stream>>>(particles,
                                                                                                   targetOffset,
                                                                                                   gForcesX,
                                                                                                   gForcesY,
                                                                                                   number_particles,
                                                                                                   gridWidth, n);
                return;
            }
        }

        if (block_width <= 4) {
            if(block_height == 256) {
                calculate_forces_two_cycles_parallel<block_width, 256><<<grid, block, 0, stream>>>(particles,
                                                                                                   targetOffset,
                                                                                                   gForcesX,
                                                                                                   gForcesY,
                                                                                                   number_particles,
                                                                                                   gridWidth, n);
                return;
            }
        }

        if (block_width <= 8) {
            if(block_height == 128) {
                calculate_forces_two_cycles_parallel<block_width, 128><<<grid, block, 0, stream>>>(particles,
                                                                                                   targetOffset,
                                                                                                   gForcesX,
                                                                                                   gForcesY,
                                                                                                   number_particles,
                                                                                                   gridWidth, n);
                return;
            }
        }

        if (block_width <= 16) {
            if (block_height == 64) {
                calculate_forces_two_cycles_parallel<block_width, 64><<<grid, block, 0, stream>>>(particles,
                                                                                                  targetOffset,
                                                                                                  gForcesX,
                                                                                                  gForcesY,
                                                                                                  number_particles,
                                                                                                  gridWidth, n);
                return;
            }
        }

        if (block_width <= 32) {
            if (block_height == 32) {
                calculate_forces_two_cycles_parallel<block_width, 32><<<grid, block, 0, stream>>>(particles,
                                                                                                  targetOffset,
                                                                                                  gForcesX,
                                                                                                  gForcesY,
                                                                                                  number_particles,
                                                                                                  gridWidth, n);
                return;
            }
        }

        if (block_width <= 64) {
            if (block_height == 16) {
                calculate_forces_two_cycles_parallel<block_width, 16><<<grid, block, 0, stream>>>(particles,
                                                                                                  targetOffset,
                                                                                                  gForcesX,
                                                                                                  gForcesY,
                                                                                                  number_particles,
                                                                                                  gridWidth, n);
                return;
            }
        }

        if (block_width <= 128) {
            if(block_height == 8) {
                calculate_forces_two_cycles_parallel<block_width, 8><<<grid, block, 0, stream>>>(particles,
                                                                                                 targetOffset,
                                                                                                 gForcesX,
                                                                                                 gForcesY,
                                                                                                 number_particles,
                                                                                                 gridWidth, n);
                return;
            }
        }

        if (block_width <= 256) {
            if (block_height == 4) {
                calculate_forces_two_cycles_parallel<block_width, 4><<<grid, block, 0, stream>>>(particles,
                                                                                                 targetOffset,
                                                                                                 gForcesX,
                                                                                                 gForcesY,
                                                                                                 number_particles,
                                                                                                 gridWidth, n);
                return;
            }
        }

        if (block_width <= 512) {
            if(block_height == 2) {
                calculate_forces_two_cycles_parallel<block_width, 2><<<grid, block, 0, stream>>>(particles,
                                                                                                 targetOffset,
                                                                                                 gForcesX,
                                                                                                 gForcesY,
                                                                                                 number_particles,
                                                                                                 gridWidth, n);
                return;
            }
        }

        if (block_width <= 1024) {
            if (block_width == 1) {
                calculate_forces_two_cycles_parallel<block_width, 1><<<grid, block, 0, stream>>>(particles,
                                                                                                 targetOffset,
                                                                                                 gForcesX,
                                                                                                 gForcesY,
                                                                                                 number_particles,
                                                                                                 gridWidth, n);
                return;
            }
        }
    }

    // Having this in a separate method for this
    // might lead to a small performance loss
    static void call_kernel_aos(
            int block_width, int block_height, particle_t *particles, int targetOffset, double *gForcesX,
            double *gForcesY, const unsigned int number_particles,
            const unsigned int gridWidth, const unsigned int n, dim3 grid, dim3 block, hipStream_t stream) {

        switch (block_width) {
            case 1024:
                call_kernel<1024>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                  number_particles, gridWidth, n, grid, block, stream);
                break;
            case 512:
                call_kernel<512>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                  number_particles, gridWidth, n, grid, block, stream);
                break;
            case 256:
                call_kernel<256>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                  number_particles, gridWidth, n, grid, block, stream);
                break;
            case 128:
                call_kernel<128>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                  number_particles, gridWidth, n, grid, block, stream);
                break;
            case 64:
                call_kernel<64>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                  number_particles, gridWidth, n, grid, block, stream);
                break;
            case 32:
                call_kernel<32>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                number_particles, gridWidth, n, grid, block, stream);
                break;
            case 16:
                call_kernel<16>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                number_particles, gridWidth, n, grid, block, stream);
                break;
            case 8:
                call_kernel<8>(block_height, particles, targetOffset,gForcesX, gForcesY,
                                number_particles, gridWidth, n, grid, block, stream);
                break;
            case 4:
                call_kernel<4>(block_height, particles, targetOffset,gForcesX, gForcesY,
                               number_particles, gridWidth, n, grid, block, stream);
                break;
            case 2:
                call_kernel<2>(block_height, particles, targetOffset,gForcesX, gForcesY,
                               number_particles, gridWidth, n, grid, block, stream);
                break;
            case 1:
                call_kernel<1>(block_height, particles, targetOffset,gForcesX, gForcesY,
                               number_particles, gridWidth, n, grid, block, stream);
                break;
        }
    }

#endif

#ifdef SOA
    void cuda_nbody_all_pairs::calculate_forces() {
        hipStream_t streams[numStreams];
        hipEvent_t events[numStreams];
        uint size = number_particles * sizeof(double);
        dim3 block(blockWidth, BLOCK_HEIGHT);

        hipMemcpy(gpu_particles_soa.x_pos, particles_soa.x_pos, size, hipMemcpyHostToDevice);
        hipMemcpy(gpu_particles_soa.y_pos, particles_soa.y_pos, size, hipMemcpyHostToDevice);

        for (int i = 0; i < numStreams; i++) {
            hipStreamCreate(&streams[i]);
            hipEventCreate(&events[i]);

            unsigned int partialHeight = (gridHeight / numStreams) +
                    (i == numStreams - 1 && (gridHeight % numStreams)) *
                    gridHeight % numStreams;
            int temp = partialHeight * BLOCK_HEIGHT;
            int targetOffset = (int)(i * BLOCK_HEIGHT * (gridHeight / numStreams));
            dim3 partialGrid(gridWidth, partialHeight);

            call_kernel_soa(
                    blockWidth, gpu_particles_soa.x_pos, gpu_particles_soa.y_pos,
                    gpu_particles_soa.mass, targetOffset, dForcesX, dForcesY,
                    number_particles, gridWidth, n, partialGrid, block);

            hipMemcpyAsync(&hForcesX[targetOffset], &dForcesX[targetOffset],
                            temp * sizeof(double),
                            hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(&hForcesY[targetOffset], &dForcesY[targetOffset],
                            temp * sizeof(double),
                            hipMemcpyDeviceToHost, streams[i]);
            hipEventRecord(events[i], streams[i]);
        }

        for (int s=0; s<numStreams; s++) {
            hipEventSynchronize(events[s]);

            unsigned int padding = (gridHeight / numStreams) +
                    (s == numStreams - 1 && (gridHeight % numStreams)) *
                    (gridHeight % numStreams);

            padding *= BLOCK_HEIGHT;
            unsigned int offset = (int)(s * BLOCK_HEIGHT * (gridHeight / numStreams));
            for (unsigned int i = offset; i < offset + padding; i++) {
                int targetParticle = i * gridWidth;
                double xF = 0; double yF = 0;
                for (int j = 0; j < gridWidth; j++) {
                    xF += hForcesX[targetParticle + j];
                    yF += hForcesY[targetParticle + j];
                }
                particles_soa.x_force[i] = xF;
                particles_soa.y_force[i] = yF;
            }
        }
    }
#else
    void cuda_nbody_all_pairs::calculate_forces() {
        hipStream_t streams[numStreams];
        hipEvent_t events[numStreams];
        uint size = number_particles * sizeof(particle_t);
        dim3 block(blockWidth, blockHeight);

        hipMemcpy(gpu_particles, particles, size, hipMemcpyHostToDevice);

        for (int i = 0; i < numStreams; i++) {
            hipStreamCreate(&streams[i]);
            hipEventCreate(&events[i]);

            unsigned int partialHeight = (gridHeight / numStreams) +
                    (i == numStreams - 1 && (gridHeight % numStreams)) *
                    gridHeight % numStreams;
            int temp = partialHeight * blockHeight;
            int targetOffset = (int)(i * blockHeight * (gridHeight / numStreams));
            dim3 partialGrid(gridWidth, partialHeight);

            call_kernel_aos(blockWidth, blockHeight,  gpu_particles, targetOffset,
                            dForcesX, dForcesY, number_particles, gridWidth,
                            n, partialGrid, block, streams[i]);

            hipMemcpyAsync(&hForcesX[targetOffset], &dForcesX[targetOffset],
                            temp * sizeof(double),
                            hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(&hForcesY[targetOffset], &dForcesY[targetOffset],
                            temp * sizeof(double),
                            hipMemcpyDeviceToHost, streams[i]);
            hipEventRecord(events[i], streams[i]);
        }

        for (int s=0; s<numStreams; s++) {
            hipEventSynchronize(events[s]);

            unsigned int padding = (gridHeight / numStreams) +
                    (s == numStreams - 1 && (gridHeight % numStreams)) *
                    (gridHeight % numStreams);

            padding *= blockHeight;
            unsigned int offset = (int)(s * blockHeight * (gridHeight / numStreams));
            for (unsigned int i = offset; i < offset + padding; i++) {
                int targetParticle = i * gridWidth;
                double xF = 0; double yF = 0;
                for (int j = 0; j < gridWidth; j++) {
                    xF += hForcesX[targetParticle + j];
                    yF += hForcesY[targetParticle + j];
                }
                particle_t *p = &particles[i];
                p->x_force = xF;
                p->y_force = yF;
            }
        }
    }
#endif

    void cuda_nbody_all_pairs::move_all_particles(double step) {
        nbody::move_all_particles(step);
    }

    void cuda_nbody_all_pairs::print_all_particles(std::ostream &out) {
        nbody::print_all_particles(out);
    }

} // namespace
