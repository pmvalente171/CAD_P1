#include "hip/hip_runtime.h"
/**
 * Hervé Paulino
 */

#include <nbody/cuda_nbody_all_pairs.h>

static constexpr int thread_block_size = 512;

namespace cadlabs {

cuda_nbody_all_pairs::cuda_nbody_all_pairs(
        const int number_particles,
        const float t_final,
        const unsigned number_of_threads,
        const universe_t universe,
        const unsigned universe_seed,
        const string file_name) :
        nbody(number_particles, t_final, universe, universe_seed, file_name),
        number_blocks ((number_particles + thread_block_size - 1)/thread_block_size)  {

    // hipMalloc((void **)&gpu_particles, number_particles*sizeof(particle_t));
}

cuda_nbody_all_pairs::~cuda_nbody_all_pairs() {
    // hipFree(gpu_particles);
}


__global__ void nbody_kernel(particle_t* particles, const unsigned number_particles) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // while (true) {}

    if (index < number_particles) {

        particle_t *pi = &particles[index];
        pi->x_force = 0;
        pi->y_force = 0;

        for (int j = 0; j < number_particles; j++) {
            particle_t *pj = &particles[j];
            /* compute the force of particle j on particle i */

            double x_sep, y_sep, dist_sq, grav_base;

            x_sep = pj->x_pos - pi->x_pos;
            y_sep = pj->y_pos - pi->y_pos;
            dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

            /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
            grav_base = GRAV_CONSTANT * (pi->mass) * (pj->mass) / dist_sq;

            pi->x_force += grav_base * x_sep;
            pi->y_force += grav_base * y_sep;
        }
    }
}


/**
 * TODO: A CUDA implementation
 */
void cuda_nbody_all_pairs::calculate_forces() {
    /* First calculate force for particles. */
    hipMalloc((void **)&gpu_particles, number_particles*sizeof(particle_t));
    uint count = number_particles * sizeof(particle_t);
    hipMemcpy(gpu_particles, particles, count, hipMemcpyHostToDevice);
    nbody_kernel<<<number_blocks, thread_block_size>>>(gpu_particles, number_particles);
    hipMemcpy(particles, gpu_particles, count, hipMemcpyDeviceToHost);
    hipFree(gpu_particles);
}


void cuda_nbody_all_pairs::move_all_particles(double step) {
    nbody::move_all_particles(step);
}

void cuda_nbody_all_pairs::print_all_particles(std::ostream &out) {
    nbody::print_all_particles(out);
}


} // namespace

